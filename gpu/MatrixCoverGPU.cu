#include "hip/hip_runtime.h"

#include "MatrixCoverGPU.cuh"

namespace gpu {

__global__ void delete_rows_and_columns(int *dl_matrix, int *deleted_rows,
                                        int *deleted_cols,
                                        const int search_depth,
                                        const int selected_row_id,
                                        const int total_dl_matrix_row_num,
                                        const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (dl_matrix[selected_row_id * total_dl_matrix_col_num + i] == 1 &&
        deleted_cols[i] == 0) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num; j++) {
        if (dl_matrix[j * total_dl_matrix_col_num + i] == 1 &&
            deleted_rows[j] == 0) {
          atomicExch(deleted_rows + j, search_depth);
        }
      }
    }
  }
}

__global__ void init_vectors(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/

__global__ void get_largest_value(int *vec, int *conflict_col_id,
                                  const int vec_length, int max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] > max) {
      max = vec[i];
    }
  }
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == max) {
      *conflict_col_id = i;
    }
  }
}

__global__ void init_vectors_reserved(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] != -1) {
      vec[i] = 0;
    }
  }
}

__global__ void check_existance_of_candidate_rows(
    int *deleted_rows, int *row_group, const int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      atomicExch(token, 1);
      atomicMin(selected_row_id, i);
    }
  }
  __syncthreads();
}

__global__ void get_vertex_row_group(int *row_group, int *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
    }
  }
}

/*
__global__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__global__ void recover_deleted_rows(int *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__global__ void recover_deleted_cols(int *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__global__ void recover_results(int *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__global__ void get_conflict_node_id(int *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}

// problem
__global__ void get_conflict_edge(int *dl_matrix, int *deleted_rows,
                                 int *deleted_cols, int *row_group, 
                                 const int conflict_node_id,
                                 const int search_depth, int *conflict_edge,
                                 const int vertex_num,
                                 const int total_dl_matrix_row_num,
                                 const int total_dl_matrix_col_num) {
  //*conflict_col_id = 0;
  //int idxa = 0;
  //int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // find the conflict edge that connects current node and the most closest
    // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    } 
    if (row_group[i] == search_depth + 1 &&
               deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge+1, i);
    }
  }
  __syncthreads();
}

__global__ void get_conflict_col_id(int *dl_matrix, int *deleted_cols, int *conflict_col_id, 
                                    int *conflict_edge, int total_dl_matrix_col_num, int vertex_num){
  //if(threadIdx.x==0){
  //  printf("conflict edge a %d edge b %d\n",conflict_edge[0],conflict_edge[1]);
 // }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num;
       j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] 
      == dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
        deleted_cols[j] > 0 && dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j]==1) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__global__ void remove_cols(int *deleted_cols, int *col_group,
                            const int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = -1;
    }
  }
}

__global__ void print_vec(int *vec, int vec_length) {
  for (int i = 0; i < vec_length; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
}

__global__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__global__ void add_gpu(int *device_arr, int device_idx, int val) {
  device_arr[device_idx] += val;
  //atomicAdd(&(device_arr[*device_idx]), val);
}

__global__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

void mc_solver(int *dl_matrix, int *results, int *deleted_cols,
               int *deleted_rows, int *col_group, int *row_group,
               int *conflict_count, const int vertex_num_gpu,
               const int total_dl_matrix_row_num_gpu,
               const int total_dl_matrix_col_num_gpu) {
  // to be refreshed if one conflict reaches many counts
  int search_depth = 0;
  int *selected_row_id_gpu;
  int vertex_num = vertex_num_gpu;
  int total_dl_matrix_col_num = total_dl_matrix_col_num_gpu;
  int total_dl_matrix_row_num = total_dl_matrix_row_num_gpu;
  int current_conflict_count;
  int *conflict_node_id_gpu;
  int *conflict_col_id_gpu;
  const int hard_conflict_threshold = 2;
  int *existance_of_candidate_rows_gpu;
  int *existance_of_candidate_rows = new int(0);
  int *conflict_col_id = new int(0);
  int *selected_row_id = new int(0);
  int *conflict_node_id = new int(0);
  int *conflict_edge = new int [2];
  int *conflict_edge_gpu;
  hipMalloc(&existance_of_candidate_rows_gpu, sizeof(int));
  hipMalloc(&selected_row_id_gpu, sizeof(int));
  hipMalloc(&conflict_node_id_gpu, sizeof(int));
  hipMalloc(&conflict_col_id_gpu, sizeof(int));
  hipMalloc(&conflict_edge_gpu, sizeof(int)*2);

  char brk;

  const int block_count = 1;
  const int thread_count = 32;
  // init lots of vectors
  init_vectors<<<block_count, thread_count>>>(conflict_count,
                                              total_dl_matrix_col_num);
  init_vectors<<<block_count, thread_count>>>(deleted_cols,
                                              total_dl_matrix_col_num);
  init_vectors<<<block_count, thread_count>>>(deleted_rows,
                                              total_dl_matrix_row_num);
  init_vectors<<<block_count, thread_count>>>(results, total_dl_matrix_row_num);
// init_vectors<<<block_count,thread_count>>>(row_group,
// total_dl_matrix_row_num);
//__syncthreads();
// get_vertex_row_group<<<block_count,thread_count >>>(row_group, dl_matrix,
// vertex_num, total_dl_matrix_row_num);
//__syncthreads();

// print_mat<<<block_count,thread_count>>>(dl_matrix, total_dl_matrix_row_num,
// total_dl_matrix_col_num);

#ifndef BENCHMARK
  print_vec<<<1, 1>>>(deleted_cols, total_dl_matrix_col_num_gpu);
  hipDeviceSynchronize();
  print_vec<<<1, 1>>>(deleted_rows, total_dl_matrix_row_num_gpu);
  hipDeviceSynchronize();
  print_vec<<<1, 1>>>(results, total_dl_matrix_row_num_gpu);
  hipDeviceSynchronize();
  print_vec<<<1, 1>>>(row_group, total_dl_matrix_row_num_gpu);
  hipDeviceSynchronize();
  print_vec<<<1, 1>>>(col_group, total_dl_matrix_col_num_gpu);
  hipDeviceSynchronize();
#endif

  for (search_depth = 1; search_depth <= vertex_num;) {

#ifndef BENCHMARK
    std::cout << "search depth is " << search_depth << std::endl;
    std::cout << "deleted_cols " << std::endl;
    print_vec<<<1, 1>>>(deleted_cols, total_dl_matrix_col_num_gpu);
    hipDeviceSynchronize();
    std::cout << "deleted_rows " << std::endl;
    print_vec<<<1, 1>>>(deleted_rows, total_dl_matrix_row_num_gpu);
    hipDeviceSynchronize();
    std::cout << "conflict count " << std::endl;
    print_vec<<<1, 1>>>(conflict_count, total_dl_matrix_col_num_gpu);
    hipDeviceSynchronize();
    std::cout << "results " << std::endl;
    print_vec<<<1, 1>>>(results, total_dl_matrix_row_num_gpu);
#endif

    hipDeviceSynchronize();
#ifndef BENCHMARK
    std::cin >> brk;
#endif
    hipMemset(existance_of_candidate_rows_gpu, 0, sizeof(int));
    hipMemset(selected_row_id_gpu, 10000, sizeof(int));
    // existance_of_candidate_rows=0;
    // selected_row_id=-1;
    check_existance_of_candidate_rows<<<block_count, thread_count>>>(
        deleted_rows, row_group, search_depth, existance_of_candidate_rows_gpu,
        selected_row_id_gpu, total_dl_matrix_row_num);
    //__syncthreads();
    hipMemcpy(existance_of_candidate_rows, existance_of_candidate_rows_gpu,
               sizeof(int), hipMemcpyDeviceToHost);

#ifndef BENCHMARK
    std::cout << "check_existance_of_candidate_rows " << *existance_of_candidate_rows <<std::endl;
#endif
    if (*existance_of_candidate_rows ==
        1) { // check if there are candidate rows existing, if no, do backtrace
      // select_row <<<block_count, thread_count >>> (deleted_rows, row_group,
      // search_depth, total_dl_matrix_row_num, selected_row_id_gpu); //select
      // row and add to results
      hipMemcpy(selected_row_id, selected_row_id_gpu, sizeof(int),
                 hipMemcpyDeviceToHost);

#ifndef BENCHMARK
      std::cout << "selected row id is " << *selected_row_id << std::endl;
#endif
      //__syncthreads();
      // hipMemset(&results[*selected_row_id],search_depth,sizeof(int));
      set_vector_value<<<1, 1>>>(results, *selected_row_id, search_depth);
      delete_rows_and_columns<<<block_count, thread_count>>>(
          dl_matrix, deleted_rows, deleted_cols, search_depth, *selected_row_id,
          total_dl_matrix_row_num,
          total_dl_matrix_col_num); // delete covered rows and columns
      //__syncthreads();
      // deleted_rows[*selected_row_id] = -search_depth;
      set_vector_value<<<1, 1>>>(deleted_rows, *selected_row_id, -search_depth);

      search_depth++; // next step
      // print_vec(deleted_cols, total_dl_matrix_col_num);
      // print_vec(deleted_rows, total_dl_matrix_row_num);
      // print_vec(conflict_count, total_dl_matrix_col_num);
      // print_vec(results, total_dl_matrix_row_num);
      continue;
    } else { // do backtrace
      hipMemset(conflict_node_id_gpu, 0, sizeof(int));
      hipMemset(conflict_col_id_gpu, 0, sizeof(int));
      init_vectors<<<1, 2>>>(conflict_edge_gpu, 2);
#ifndef BENCHMARK
      std::cout<<"search depth = "<< search_depth << std::endl;
#endif
      search_depth--;
#ifndef BENCHMARK
      std::cout<<"search depth = "<< search_depth << std::endl;
#endif
      if (search_depth > 0) {
        // conflict_node_id = get_conflict_node_id(deleted_rows, row_group,
        // search_depth, total_dl_matrix_row_num);
        get_conflict_node_id<<<block_count, thread_count>>>(
            deleted_rows, row_group, search_depth, conflict_node_id_gpu,
            total_dl_matrix_row_num);
        hipMemcpy(conflict_node_id, conflict_node_id_gpu, sizeof(int),
                   hipMemcpyDeviceToHost);


        get_conflict_edge<<<block_count, thread_count>>>(
            dl_matrix, deleted_rows, deleted_cols, row_group, *conflict_node_id,
            search_depth, conflict_edge_gpu, vertex_num,
            total_dl_matrix_row_num, total_dl_matrix_col_num);
        
        hipMemcpy(conflict_edge, conflict_edge_gpu, sizeof(int)*2,
            hipMemcpyDeviceToHost);

        get_conflict_col_id<<<block_count, thread_count>>>(
            dl_matrix, deleted_cols, conflict_col_id_gpu, 
            conflict_edge_gpu, total_dl_matrix_col_num, vertex_num);

        hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
                   hipMemcpyDeviceToHost);
        
#ifndef BENCHMARK
        std::cout<<"conflict node id is "<<*conflict_node_id<<std::endl;

        std::cout<<"conflict col id is "<<*conflict_col_id<<std::endl;
        if(*conflict_col_id==0){
          std::cout<<"conflict edge a is "<<conflict_edge[0]<<std::endl;
          std::cout<<"conflict edge b is "<<conflict_edge[1]<<std::endl;
          hipDeviceSynchronize();
          std::cout << "row 1 " << std::endl;
          print_vec<<<1, 1>>>(dl_matrix+conflict_edge[0]*total_dl_matrix_col_num_gpu, total_dl_matrix_col_num_gpu);
          hipDeviceSynchronize();
          std::cout << "row 2 " << std::endl;
          print_vec<<<1, 1>>>(dl_matrix+conflict_edge[1]*total_dl_matrix_col_num_gpu, total_dl_matrix_col_num_gpu);
          hipDeviceSynchronize();
        }
#endif

        // conflict_count[*conflict_col_id]++; //update conflict edge count
        add_gpu<<<1, 1>>>(conflict_count, *conflict_col_id, 1);
        recover_deleted_rows<<<block_count, thread_count>>>(
            deleted_rows, search_depth,
            total_dl_matrix_row_num); // recover deleted rows  previously
                                      // selected rows
        //__syncthreads();
        recover_deleted_cols<<<block_count, thread_count>>>(
            deleted_cols, search_depth,
            total_dl_matrix_col_num); // recover deleted cols except afftected
                                      // by previously selected rows
        //__syncthreads();
        recover_results<<<block_count, thread_count>>>(
            results, search_depth, total_dl_matrix_row_num); // recover results
        //__syncthreads();
        hipMemcpy(&current_conflict_count, &conflict_count[*conflict_col_id],
                   sizeof(int), hipMemcpyDeviceToHost);
        if (current_conflict_count > hard_conflict_threshold) {
          search_depth = 1;
          init_vectors<<<block_count, thread_count>>>(conflict_count,
                                                      total_dl_matrix_col_num);
          init_vectors_reserved<<<block_count, thread_count>>>(
              deleted_cols, total_dl_matrix_col_num);
          init_vectors<<<block_count, thread_count>>>(deleted_rows,
                                                      total_dl_matrix_row_num);
          init_vectors<<<block_count, thread_count>>>(results,
                                                      total_dl_matrix_row_num);
          //__syncthreads();
          remove_cols<<<block_count, thread_count>>>(deleted_cols, col_group,
                                                     *conflict_col_id,
                                                     total_dl_matrix_col_num);

          //__syncthreads();
          // deleted_cols[*conflict_col_id] = -1;
          hipMemset(&deleted_cols[*conflict_col_id], -1, sizeof(int));
          continue;
        }
      } else { // if all vertices are gone through, directly remove the edge
               // with largest conflict count.
#ifndef BENCHMARK
        std::cout<<"reset state"<<std::endl;
        std::cout<<"======================================================================================"<<std::endl;
#endif
        search_depth = 1;
        get_largest_value<<<block_count, thread_count>>>(
            conflict_count, conflict_col_id_gpu, total_dl_matrix_col_num, 0);
        hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
                   hipMemcpyDeviceToHost);
        //__syncthreads();
        init_vectors<<<block_count, thread_count>>>(conflict_count,
                                                    total_dl_matrix_col_num);
        init_vectors_reserved<<<block_count, thread_count>>>(
            deleted_cols, total_dl_matrix_col_num);
        init_vectors<<<block_count, thread_count>>>(deleted_rows,
                                                    total_dl_matrix_row_num);
        init_vectors<<<block_count, thread_count>>>(results,
                                                    total_dl_matrix_row_num);
        //__syncthreads();
        remove_cols<<<block_count, thread_count>>>(
            deleted_cols, col_group, *conflict_col_id, total_dl_matrix_col_num);
        continue;
      }
      // print_vec(deleted_cols, total_dl_matrix_col_num);
      // print_vec(deleted_rows, total_dl_matrix_row_num);
      // print_vec(conflict_count, total_dl_matrix_col_num);
      // print_vec(results, total_dl_matrix_row_num);
    }
  }

  hipFree(existance_of_candidate_rows_gpu);
  hipFree(selected_row_id_gpu);
  hipFree(conflict_col_id_gpu);
  hipFree(conflict_node_id_gpu);
  hipFree(conflict_edge);
  delete existance_of_candidate_rows;
  delete conflict_col_id;
  delete selected_row_id;
  delete conflict_node_id;
  delete [] conflict_edge;
}

} // namespace gpu

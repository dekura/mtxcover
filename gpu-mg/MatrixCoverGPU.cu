#include "hip/hip_runtime.h"
#include "MatrixCoverGPU.cuh"

namespace gpu_mg {

constexpr int size_bit = 1 << 31;

__device__ void delete_rows_and_columns(
    const int *dl_matrix, const int *next_row, const int *next_col,
    int *deleted_rows, int *deleted_cols, const int search_depth,
    const int selected_row_id, const int total_dl_matrix_row_num,
    const int total_dl_matrix_col_num) {
  int selected_row_idx = selected_row_id * total_dl_matrix_col_num;

  for (int i = threadIdx.x; i < total_dl_matrix_col_num;
       // // The below line will have negative effect of the col number is small
       //  i += (next_col[selected_row_idx + i] + blockDim.x - 1) / blockDim.x
       i += blockDim.x) {
    if (deleted_cols[i] == 0 && dl_matrix[selected_row_idx + i] == 1) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num;
           j += next_row[i * total_dl_matrix_row_num + j]) {
        if (deleted_rows[j] == 0 &&
            dl_matrix[j * total_dl_matrix_col_num + i] == 1) {
          deleted_rows[j] = search_depth;
        }
      }
    }
  }
  __syncthreads();
}

__device__ void init_vectors(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/

__device__ void get_largest_value(int *vec, const int vec_length, int *max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {

    atomicMax(max, vec[i]);
  }
}

__device__ void find_index(int *vec, const int vec_length, int *value,
                           int *index) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == *value) {
      atomicMax(index, i);
    }
  }
}

__device__ void init_vectors_reserved(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    // if (vec[i] != -1) {
    vec[i] &= size_bit;
    // }
  }
}

__device__ void check_existance_of_candidate_rows(
    int *deleted_rows, int *row_group, const int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i<total_dl_matrix_row_num && * selected_row_id> i;
       i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      // atomicExch(token, 1);
      *token = 1;
      atomicMin(selected_row_id, i);
      // If find a number can break;
      break;
    }
  }
  __syncthreads();
}

__device__ void get_vertex_row_group(int *row_group, int *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
    }
  }
}

/*
__device__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__device__ void recover_deleted_rows(int *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__device__ void recover_deleted_cols(int *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__device__ void recover_results(int *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__device__ void get_conflict_node_id(int *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] < search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_edge(int *dl_matrix, int *deleted_rows,
                                  int *row_group, const int conflict_node_id,
                                  const int search_depth, int *conflict_edge,
                                  const int vertex_num,
                                  const int total_dl_matrix_row_num,
                                  const int total_dl_matrix_col_num) {
  //*conflict_col_id = 0;
  // int idxa = 0;
  // int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // find the conflict edge that connects current node and the most closest
    // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    }
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge + 1, i);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_col_id(int *dl_matrix, int *deleted_cols,
                                    int *conflict_col_id, int *conflict_edge,
                                    int total_dl_matrix_col_num,
                                    int vertex_num) {
  // if(threadIdx.x==0){
  //  printf("conflict edge a %d edge b
  //  %d\n",conflict_edge[0],conflict_edge[1]);
  // }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num; j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] ==
            dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
        deleted_cols[j] > 0 &&
        dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] == 1) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__device__ void remove_cols(int *deleted_cols, int *col_group,
                            const int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = size_bit;
    }
  }
}

__device__ void print_vec(int *vec, int vec_length) {
  for (int i = 0; i < vec_length; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
}

/*
__global__ inline void print_vec_g(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}

*/
__device__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__device__ void add_gpu(int *device_var, int val) {
  atomicAdd(device_var, val);
}

__device__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

__global__ void
mc_solver(int *dl_matrix, int *next_col, int *next_row, int *results,
          int *deleted_cols, int *deleted_rows, int *col_group, int *row_group,
          int *conflict_count, int *vertex_num, int *total_dl_matrix_row_num,
          int *total_dl_matrix_col_num, int *offset_col, int *offset_row,
          int *offset_matrix, int *search_depth, int *selected_row_id,
          int *current_conflict_count, int *conflict_node_id,
          int *conflict_col_id, int *existance_of_candidate_rows,
          int *conflict_edge, int *max, const int graph_count,
          const int hard_conflict_threshold) {
  // // The Max Col Cnt is 131
  // // The Max Row Cnt is 339
  __shared__ int shared_deleted_cols[140];
  __shared__ int shared_deleted_rows[340];
  __shared__ int shared_conflict_count[140];
  __shared__ int shared_result[340];
  __shared__ int shared_row_group[340];
  __shared__ int shared_col_group[140];
  __shared__ int shared_max;
  __shared__ int shared_existance_of_candidate_rows;
  __shared__ int shared_selected_row_id;
  __shared__ int shared_conflict_node_id;
  __shared__ int shared_conflict_col_id;
  __shared__ int shared_conflict_edge[2];

  for (int k = blockIdx.x; k < graph_count; k += gridDim.x) {
    int vertex_cn = vertex_num[k];
    int t_cn = total_dl_matrix_col_num[k];
    int t_rn = total_dl_matrix_row_num[k];
    int *t_conflict_count = conflict_count + offset_col[k];
    int *t_deleted_cols = deleted_cols + offset_col[k];
    int *t_deleted_rows = deleted_rows + offset_row[k];
    int *t_results = results + offset_row[k];
    int *t_row_group = row_group + offset_row[k];
    int *t_col_group = col_group + offset_col[k];
    int *t_dl_matrix = dl_matrix + offset_matrix[k];
    int *t_next_col = next_col + offset_matrix[k];
    int *t_next_row = next_row + offset_matrix[k];
    int *t_conflict_edge = conflict_edge + 2 * k;
    int local_search_depth = 1;

#ifndef BENCHMARK
    printf("blockID is %d\n", k);
    printf("vertexnum is %d\n", vertex_num[k]);
    printf("init conflict count \n");
#endif
    init_vectors(t_conflict_count, t_cn);
#ifndef BENCHMARK
    for (int i = 0; i < t_cn; i++) {
      printf("%d ", t_conflict_count[i]);
    }
    printf("\n");
#endif
    init_vectors(t_deleted_cols, t_cn);
    init_vectors(t_deleted_rows, t_rn);
    init_vectors(t_results, t_rn);
    __syncthreads();
    get_vertex_row_group(t_row_group, t_dl_matrix, vertex_num[k], t_rn, t_cn);
    __syncthreads();

    for (int i = threadIdx.x; i < t_cn; i += blockDim.x) {
      shared_deleted_cols[i] = t_deleted_cols[i];
      shared_conflict_count[i] = t_conflict_count[i];
      shared_col_group[i] = t_col_group[i];
    }
    for (int i = threadIdx.x; i < t_rn; i += blockDim.x) {
      shared_deleted_rows[i] = t_deleted_rows[i];
      shared_result[i] = t_results[i];
      shared_row_group[i] = t_row_group[i];
    }
    shared_max = 0;
    __syncthreads();

    /*
    print_vec(deleted_cols+offset_col[k], t_cn);
    __syncthreads();
    print_vec(deleted_rows+offset_row[k], t_rn);
    __syncthreads();
    print_vec(results+offset_row[k], t_rn);
    __syncthreads();
    print_vec(row_group+offset_row[k], t_rn);
    __syncthreads();
    print_vec(col_group+offset_col[k], t_cn);
    __syncthreads();
    */

    for (local_search_depth = 1; local_search_depth <= vertex_cn;) {

#ifndef BENCHMARK
      printf("search depth is %d\n", local_search_depth);
      // std::cout<<"deleted_cols "<<std::endl;
      // hipDeviceSynchronize();
      printf("deleted_cols\n");
      print_vec(shared_deleted_cols, t_cn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("deleted_rows\n");
      print_vec(shared_deleted_rows, t_rn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("results\n");
      print_vec(t_results, t_rn);
// hipDeviceSynchronize();
#endif

      shared_existance_of_candidate_rows = 0;
      shared_selected_row_id = t_rn;
      shared_conflict_node_id = 0;
      shared_conflict_col_id = 0;
      shared_conflict_edge[0] = 0;
      shared_conflict_edge[1] = 0;
      check_existance_of_candidate_rows(
          shared_deleted_rows, shared_row_group, local_search_depth,
          &shared_existance_of_candidate_rows, &shared_selected_row_id, t_rn);
      __syncthreads();
      if (shared_existance_of_candidate_rows ==
          1) { // check if there are candidate
               // rows existing, if no, do
               // backtrace
// select_row <<<block_count, thread_count >>> (deleted_rows, row_group,
// search_depth, total_dl_matrix_row_num, selected_row_id_gpu); //select
// row and add to results
// hipMemcpy(selected_row_id, selected_row_id_gpu, sizeof(int),
// hipMemcpyDeviceToHost);
#ifndef BENCHMARK
        printf("selected row id is %d \n", shared_selected_row_id);
#endif
        shared_result[shared_selected_row_id] = local_search_depth;
        delete_rows_and_columns(t_dl_matrix, t_next_row, t_next_col,
                                shared_deleted_rows, shared_deleted_cols,
                                local_search_depth, shared_selected_row_id,
                                t_rn,
                                t_cn); // delete covered rows and columns
        __syncthreads();
        shared_deleted_rows[shared_selected_row_id] = -local_search_depth;
        local_search_depth++; // next step
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      } else { // do backtrace
        local_search_depth--;
        if (local_search_depth > 0) {

          get_conflict_node_id(shared_deleted_rows, shared_row_group,
                               local_search_depth, &shared_conflict_node_id,
                               t_rn);
          if (shared_conflict_node_id > 0) {
            __syncthreads();
            get_conflict_edge(t_dl_matrix, shared_deleted_rows,
                              shared_row_group, shared_conflict_node_id,
                              local_search_depth, shared_conflict_edge,
                              vertex_num[k], t_rn, t_cn);
            __syncthreads();
            get_conflict_col_id(t_dl_matrix, shared_deleted_cols,
                                &shared_conflict_col_id, shared_conflict_edge,
                                t_cn, vertex_num[k]);
            __syncthreads();

            // update conflict edge count
            shared_conflict_count[shared_conflict_col_id]++;
            recover_deleted_rows(shared_deleted_rows, local_search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(shared_deleted_cols, local_search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(shared_result, local_search_depth,
                            t_rn); // recover results
            //__syncthreads();
            if (shared_conflict_count[shared_conflict_col_id] >
                hard_conflict_threshold) {
              local_search_depth = 1;
              init_vectors(shared_conflict_count, t_cn);
              init_vectors_reserved(shared_deleted_cols, t_cn);
              init_vectors(shared_deleted_rows, t_rn);
              init_vectors(shared_result, t_rn);
              __syncthreads();
              remove_cols(shared_deleted_cols, shared_col_group,
                          shared_conflict_col_id, t_cn);
              __syncthreads();
              shared_deleted_cols[shared_conflict_col_id] = size_bit;
            }
          } else {
            recover_deleted_rows(shared_deleted_rows, local_search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(shared_deleted_cols, local_search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(shared_result, local_search_depth,
                            t_rn); // recover results
          }
        } else { // if all vertices are gone through, directly remove the edge
                 // with largest conflict count.
          local_search_depth = 1;
          get_largest_value(shared_conflict_count, t_cn, &shared_max);

          __syncthreads();
          find_index(shared_conflict_count, t_cn, &shared_max,
                     &shared_conflict_col_id);
          init_vectors(shared_conflict_count, t_cn);
          init_vectors_reserved(shared_deleted_cols, t_cn);
          init_vectors(shared_deleted_rows, t_rn);
          init_vectors(shared_result, t_rn);
          __syncthreads();
          remove_cols(shared_deleted_cols, shared_col_group,
                      shared_conflict_col_id, t_cn);
        }
      }
    }

    max[k] = shared_max;
    search_depth[k] = local_search_depth;
    existance_of_candidate_rows[k] = shared_existance_of_candidate_rows;
    selected_row_id[k] = shared_selected_row_id;
    conflict_node_id[k] = shared_conflict_node_id;
    conflict_col_id[k] = shared_conflict_col_id;
    t_conflict_edge[0] = shared_conflict_edge[0];
    t_conflict_edge[1] = shared_conflict_edge[1];
    for (int i = threadIdx.x; i < t_cn; i += blockDim.x) {
      t_deleted_cols[i] = shared_deleted_cols[i];
      t_conflict_count[i] = shared_conflict_count[i];
      t_col_group[i] = shared_col_group[i];
    }
    for (int i = threadIdx.x; i < t_rn; i += blockDim.x) {
      t_deleted_rows[i] = shared_deleted_rows[i];
      t_results[i] = shared_result[i];
      t_row_group[i] = shared_row_group[i];
    }
    __syncthreads();
  }
}

} // namespace gpu_mg

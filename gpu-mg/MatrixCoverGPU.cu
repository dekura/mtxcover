#include "hip/hip_runtime.h"
#include "MatrixCoverGPU.cuh"

namespace gpu_mg {

constexpr int size_bit = 1 << 31;

__device__ void delete_rows_and_columns(int *dl_matrix, int *deleted_rows,
                                        int *deleted_cols,
                                        const int search_depth,
                                        const int selected_row_id,
                                        const int total_dl_matrix_row_num,
                                        const int total_dl_matrix_col_num) {
  int selected_row_idx = selected_row_id * total_dl_matrix_col_num;

  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] == 0 && dl_matrix[selected_row_idx + i] == 1) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num; ++j) {
        if (deleted_rows[j] == 0 &&
            dl_matrix[j * total_dl_matrix_col_num + i] == 1) {
          deleted_rows[j] = search_depth;
        }
      }
    }
  }
  __syncthreads();
}

__device__ void init_vectors(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/

__device__ void get_largest_value(int *vec, int *conflict_col_id,
                                  const int vec_length, int max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] > max) {
      max = vec[i];
    }
    // atomicMax(&max, vec[i]);
  }
  __syncthreads();
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == max) {
      *conflict_col_id = i;
      break;
    }
  }
}

__device__ void init_vectors_reserved(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    // if (vec[i] != -1) {
    vec[i] &= size_bit;
    // }
  }
}

__device__ void check_existance_of_candidate_rows(
    int *deleted_rows, int *row_group, const int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i<total_dl_matrix_row_num && * selected_row_id> i;
       i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      // atomicExch(token, 1);
      *token = 1;
      atomicMin(selected_row_id, i);
      // If find a number can break;
      break;
    }
  }
  __syncthreads();
}

__device__ void get_vertex_row_group(int *row_group, int *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
    }
  }
}

/*
__device__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__device__ void recover_deleted_rows(int *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__device__ void recover_deleted_cols(int *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__device__ void recover_results(int *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__device__ void get_conflict_node_id(int *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}


__device__ void get_conflict_edge(int *dl_matrix, int *deleted_rows,
                                  int *row_group, 
                                  const int conflict_node_id,
                                  const int search_depth, int *conflict_edge,
                                  const int vertex_num,
                                  const int total_dl_matrix_row_num,
                                  const int total_dl_matrix_col_num) {
//*conflict_col_id = 0;
//int idxa = 0;
//int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
  // find the conflict edge that connects current node and the most closest
  // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    } 
    if (row_group[i] == search_depth + 1 &&
      deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge+1, i);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_col_id(int *dl_matrix, int *deleted_cols, 
  int *conflict_col_id, int *conflict_edge, 
  int total_dl_matrix_col_num, int vertex_num){
//if(threadIdx.x==0){
//  printf("conflict edge a %d edge b %d\n",conflict_edge[0],conflict_edge[1]);
// }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num;
  j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] 
    == dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
    deleted_cols[j] > 0 && dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j]==1) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__device__ void remove_cols(int *deleted_cols, int *col_group,
                            const int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = size_bit;
    }
  }
}


__device__ void print_vec(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}


/*
__global__ inline void print_vec_g(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}

*/
__device__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__device__ void add_gpu(int *device_var, int val) {
  atomicAdd(device_var, val);
}

__device__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

__global__ void
mc_solver(int *dl_matrix, int *next_col, int *next_row, int *results,
          int *deleted_cols, int *deleted_rows, int *col_group, int *row_group,
          int *conflict_count, int *vertex_num, int *total_dl_matrix_row_num,
          int *total_dl_matrix_col_num, int *offset_col, int *offset_row,
          int *offset_matrix, int *search_depth, int *selected_row_id,
          int *current_conflict_count, int *conflict_node_id,
          int *conflict_col_id, int *existance_of_candidate_rows, int* conflict_edge,
          const int graph_count, const int hard_conflict_threshold) {

  // to be refreshed if one conflict reaches many counts
  /*
  int search_depth = 0;
  int *selected_row_id_gpu;
  int vertex_num = vertex_num_gpu;
  int total_dl_matrix_col_num=total_dl_matrix_col_num_gpu;
  int total_dl_matrix_row_num=total_dl_matrix_row_num_gpu;
  int current_conflict_count;
  int *conflict_node_id_gpu;
  int *conflict_col_id_gpu;
  const int hard_conflict_threshold = 500;
  int *existance_of_candidate_rows_gpu;
  int *existance_of_candidate_rows=new int(0);
  int *conflict_col_id=new int(0);
  int *selected_row_id=new int(0);
  int *conflict_node_id=new int(0);
  hipMalloc(&existance_of_candidate_rows_gpu, sizeof(int));
  hipMalloc(&selected_row_id_gpu, sizeof(int));
  hipMalloc(&conflict_node_id_gpu, sizeof(int));
  hipMalloc(&conflict_col_id_gpu, sizeof(int));

  char brk;
  */
  // int k = blockDim.x;
  for (int k = blockIdx.x; k < graph_count; k += gridDim.x) {
    int t_cn = total_dl_matrix_col_num[k];
    int t_rn = total_dl_matrix_row_num[k];
    int *t_conflict_count = conflict_count + offset_col[k];
    int *t_deleted_cols = deleted_cols + offset_col[k];
    int *t_deleted_rows = deleted_rows + offset_row[k];
    int *t_results = results + offset_row[k];
    int *t_row_group = row_group + offset_row[k];
    int *t_col_group = col_group + offset_col[k];
    int *t_dl_matrix = dl_matrix + offset_matrix[k];
    int *t_next_col = next_col + offset_matrix[k];
    int *t_next_row = next_row + offset_matrix[k];
    int *t_conflict_edge = conflict_edge + 2*k;

#ifdef BENCHMARK
    printf("blockID is %d\n", k);
    printf("vertexnum is %d\n", vertex_num[k]);
    printf("init conflict count \n");
#endif
    init_vectors(t_conflict_count, t_cn);
#ifdef BENCHMARK
    for (int i = 0; i < t_cn; i++) {
      printf("%d ", t_conflict_count[i]);
    }
    printf("\n");
#endif
    init_vectors(t_deleted_cols, t_cn);
    init_vectors(t_deleted_rows, t_rn);
    init_vectors(t_results, t_rn);
    __syncthreads();
    get_vertex_row_group(t_row_group, t_dl_matrix, vertex_num[k], t_rn, t_cn);
    __syncthreads();
    /*
    print_vec(deleted_cols+offset_col[k], t_cn);
    __syncthreads();
    print_vec(deleted_rows+offset_row[k], t_rn);
    __syncthreads();
    print_vec(results+offset_row[k], t_rn);
    __syncthreads();
    print_vec(row_group+offset_row[k], t_rn);
    __syncthreads();
    print_vec(col_group+offset_col[k], t_cn);
    __syncthreads();
    */

    for (search_depth[k] = 1; search_depth[k] <= vertex_num[k];) {
#ifdef BENCHMARK
      printf("search depth is %d\n", search_depth[k]);
      // std::cout<<"deleted_cols "<<std::endl;
      // hipDeviceSynchronize();
      printf("deleted_cols\n");
      print_vec(t_deleted_cols, t_cn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("deleted_rows\n");
      print_vec(t_deleted_rows, t_rn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("results\n");
      print_vec(t_results, t_rn);
// hipDeviceSynchronize();
#endif

      existance_of_candidate_rows[k] = 0;
      selected_row_id[k] = t_rn;
      conflict_node_id[k] = 0;
      conflict_col_id[k] = 0;
      t_conflict_edge[0] = 0;
      t_conflict_edge[1] = 0;
      // existance_of_candidate_rows=0;
      // selected_row_id=-1;
      check_existance_of_candidate_rows(
          t_deleted_rows, t_row_group, search_depth[k],
          existance_of_candidate_rows + k, selected_row_id + k, t_rn);
      __syncthreads();
      // printf()
      // hipMemcpy(existance_of_candidate_rows,
      // existance_of_candidate_rows_gpu, sizeof(int), hipMemcpyDeviceToHost);
      // std::cout<<"check_existance_of_candidate_rows "<<std::endl;
      if (existance_of_candidate_rows[k] == 1) { // check if there are candidate
                                                 // rows existing, if no, do
                                                 // backtrace
// select_row <<<block_count, thread_count >>> (deleted_rows, row_group,
// search_depth, total_dl_matrix_row_num, selected_row_id_gpu); //select
// row and add to results
// hipMemcpy(selected_row_id, selected_row_id_gpu, sizeof(int),
// hipMemcpyDeviceToHost);
#ifdef BENCHMARK
        printf("selected row id is %d \n", selected_row_id[k]);
#endif
        //__syncthreads();
        // hipMemset(&results[*selected_row_id],search_depth,sizeof(int));
        t_results[selected_row_id[k]] = search_depth[k];
        // set_vector_value<<<1,1>>>(results, *selected_row_id, search_depth);
        delete_rows_and_columns(t_dl_matrix, t_deleted_rows, t_deleted_cols,
                                search_depth[k], selected_row_id[k], t_rn,
                                t_cn); // delete covered rows and columns
        __syncthreads();
        // deleted_rows[*selected_row_id] = -search_depth;
        t_deleted_rows[selected_row_id[k]] = -search_depth[k];
        // set_vector_value<<<1,1>>>(deleted_rows, *selected_row_id,
        // -search_depth);

        search_depth[k]++; // next step
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      } else { // do backtrace
        search_depth[k]--;
        if (search_depth[k] > 0) {

          get_conflict_node_id(t_deleted_rows, t_row_group, search_depth[k],
                               conflict_node_id + k, t_rn);
          __syncthreads();
          get_conflict_edge(t_dl_matrix, t_deleted_rows,t_row_group, 
                                conflict_node_id[k],
                                search_depth[k], t_conflict_edge,
                                 vertex_num[k], t_rn, t_cn);
          __syncthreads();
          get_conflict_col_id(t_dl_matrix, t_deleted_cols, conflict_col_id + k,
                            t_conflict_edge, t_cn, vertex_num[k]);
          __syncthreads();

          // conflict_count[*conflict_col_id]++;
          // update conflict edge count
          t_conflict_count[conflict_col_id[k]]++;
          // add_gpu<<<1,1>>>(&deleted_rows[*selected_row_id],1);
          recover_deleted_rows(t_deleted_rows, search_depth[k],
                               t_rn); // recover deleted
                                      // rows  previously
                                      // selected rows
          __syncthreads();
          recover_deleted_cols(t_deleted_cols, search_depth[k],
                               t_cn); // recover deleted
                                      // cols except
                                      // afftected by
                                      // previously
                                      // selected rows
          __syncthreads();
          recover_results(t_results, search_depth[k], t_rn); // recover results
          //__syncthreads();
          // hipMemcpy(&current_conflict_count,
          // &conflict_count[*conflict_col_id], sizeof(int),
          // hipMemcpyDeviceToHost);
          if (t_conflict_count[conflict_col_id[k]] > hard_conflict_threshold) {
            search_depth[k] = 1;
            init_vectors(t_conflict_count, t_cn);
            init_vectors_reserved(t_deleted_cols, t_cn);
            init_vectors(t_deleted_rows, t_rn);
            init_vectors(t_results, t_rn);
            __syncthreads();
            remove_cols(t_deleted_cols, t_col_group, conflict_col_id[k], t_cn);
            __syncthreads();
            t_deleted_cols[conflict_col_id[k]] = size_bit;
            continue;
            // /hipMemset(&deleted_cols[*conflict_col_id],-1,sizeof(int));
          }
        } else { // if all vertices are gone through, directly remove the edge
                 // with largest conflict count.
          search_depth[k] = 1;
          get_largest_value(t_conflict_count, conflict_col_id + k, t_cn, 0);
          // hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
          // hipMemcpyDeviceToHost);
          __syncthreads();
          init_vectors(t_conflict_count, t_cn);
          init_vectors_reserved(t_deleted_cols, t_cn);
          init_vectors(t_deleted_rows, t_rn);
          init_vectors(t_results, t_rn);
          __syncthreads();
          remove_cols(t_deleted_cols, t_col_group, conflict_col_id[k], t_cn);
          continue;
        }
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      }
    }
  }
}

} // namespace gpu_mg

#include "hip/hip_runtime.h"
#include "MatrixCoverGPU.cuh"

namespace gpu_mg {

//constexpr int size_bit = 1 << 31;

__device__ void delete_rows_and_columns(
    const int *dl_matrix, const int *next_row, const int *next_col,
    short *deleted_rows, short *deleted_cols, const int search_depth,
    const int selected_row_id, const int total_dl_matrix_row_num,
    const int total_dl_matrix_col_num) {
  int selected_row_idx = selected_row_id * total_dl_matrix_col_num;

  for (int i = threadIdx.x; i < total_dl_matrix_col_num;
       // // The below line will have negative effect of the col number is small
       //  i += (next_col[selected_row_idx + i] + blockDim.x - 1) / blockDim.x
       i += blockDim.x) {
    if (deleted_cols[i] == 0 && dl_matrix[selected_row_idx + i] == 1) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num;
           j += next_row[i * total_dl_matrix_row_num + j]) {
        if (deleted_rows[j] == 0 &&
            dl_matrix[j * total_dl_matrix_col_num + i] == 1) {
          deleted_rows[j] = search_depth;
        }
      }
    }
  }
  __syncthreads();
}

__device__ void init_vectors(short *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/

__device__ void get_largest_value(short *vec, const int vec_length, int *max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {

    atomicMax(max, vec[i]);
  }
}

__device__ void find_index(short *vec, const int vec_length, int *value,
                           int *index) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == *value) {
      atomicMax(index, i);
    }
  }
}

__device__ void init_vectors_reserved(short *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] != -1) {
      vec[i] = 0;
    }
  }
}

__device__ void check_existance_of_candidate_rows(
    short *deleted_rows, int *row_group, const int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i<total_dl_matrix_row_num && * selected_row_id> i;
       i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      // atomicExch(token, 1);
      *token = 1;
      atomicMin(selected_row_id, i);
      // If find a number can break;
      break;
    }
  }
  __syncthreads();
}

__device__ void get_vertex_row_group(int *row_group, int *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
    }
  }
}

/*
__device__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__device__ void recover_deleted_rows(short *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__device__ void recover_deleted_cols(short *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__device__ void recover_results(short *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__device__ void get_conflict_node_id(short *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] < search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_edge(int *dl_matrix, short *deleted_rows,
                                  int *row_group, const int conflict_node_id,
                                  const int search_depth, int *conflict_edge,
                                  const int vertex_num,
                                  const int total_dl_matrix_row_num,
                                  const int total_dl_matrix_col_num) {
  //*conflict_col_id = 0;
  // int idxa = 0;
  // int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // find the conflict edge that connects current node and the most closest
    // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    }
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge + 1, i);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_col_id(int *dl_matrix, short *deleted_cols,
                                    int *conflict_col_id, int *conflict_edge,
                                    int total_dl_matrix_col_num,
                                    int vertex_num) {
  // if(threadIdx.x==0){
  //  printf("conflict edge a %d edge b
  //  %d\n",conflict_edge[0],conflict_edge[1]);
  // }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num; j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] ==
            dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
        deleted_cols[j] > 0 &&
        dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] == 1) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__device__ void remove_cols(short *deleted_cols, int *col_group,
                            const int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = -1;
    }
  }
}

__device__ void print_vec(int *vec, int vec_length) {
  for (int i = 0; i < vec_length; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
}

/*
__global__ inline void print_vec_g(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}

*/
__device__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__device__ void add_gpu(int *device_var, int val) {
  atomicAdd(device_var, val);
}

__device__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

__global__ void 
init_vertex_group(int *row_group, int *dl_matrix, int* vertex_num, int* t_cn, int* t_rn, int *offset_row, int *offset_matrix, int graph_count) {
    int k=blockIdx.x;
    if(k<graph_count){
        get_vertex_row_group(row_group+offset_row[k], dl_matrix+offset_matrix[k], vertex_num[k], t_rn[k], t_cn[k]);
    }

}

__global__ void
mc_solver(int *dl_matrix, int *next_col, int *next_row, int *results,
          int *_deleted_cols, int *_deleted_rows, int *col_group, int *row_group,
          int *conflict_count, int *vertex_num, int *total_dl_matrix_row_num,
          int *total_dl_matrix_col_num, int *offset_col, int *offset_row,
          int *offset_matrix, int *_search_depth, int *selected_row_id,
          int *_current_conflict_count, int *_conflict_node_id,
          int *_conflict_col_id, int *_existance_of_candidate_rows,
          int *_conflict_edge, int *_max, const int graph_count,
          const int hard_conflict_threshold) {



  //add shared mem

  __shared__ short t_deleted_rows[256];
  __shared__ short t_deleted_cols[128];
  __shared__ short t_conflict_count[256];
  __shared__ short t_results[256];
  __shared__ int t_conflict_edge[2];
  __shared__ short search_depth;
  __shared__ int t_max;
  __shared__ int t_existance_of_candidate_rows;
  __shared__ int t_conflict_node_id;
  __shared__ int t_conflict_col_id;
  //__shared__
  //end add shared mem

  int k = blockIdx.x;
  if(k<graph_count){
  //for (int k = blockIdx.x; k < graph_count; k += gridDim.x) {
    int t_cn = total_dl_matrix_col_num[k];
    int t_rn = total_dl_matrix_row_num[k];
    //int *t_conflict_count = conflict_count + offset_col[k];
    //int *t_deleted_cols = deleted_cols + offset_col[k];
    //int *t_deleted_rows = deleted_rows + offset_row[k];
    int *t_final_results = results + offset_row[k];
    int *t_row_group = row_group + offset_row[k];
    int *t_col_group = col_group + offset_col[k];
    int *t_dl_matrix = dl_matrix + offset_matrix[k];
    int *t_next_col = next_col + offset_matrix[k];
    int *t_next_row = next_row + offset_matrix[k];
    //int *t_conflict_edge = conflict_edge + 2 * k;
    

#ifndef BENCHMARK
    printf("blockID is %d\n", k);
    printf("vertexnum is %d\n", vertex_num[k]);
    printf("init conflict count \n");
#endif
    init_vectors(t_conflict_count, t_cn);
#ifndef BENCHMARK
    for (int i = 0; i < t_cn; i++) {
      printf("%d ", t_conflict_count[i]);
    }
    printf("\n");
#endif
    init_vectors(t_deleted_cols, t_cn);
    init_vectors(t_deleted_rows, t_rn);
    init_vectors(t_results, t_rn);
    __syncthreads();
    //get_vertex_row_group(t_row_group, t_dl_matrix, vertex_num[k], t_rn, t_cn);
    //__syncthreads();
    /*
    print_vec(deleted_cols+offset_col[k], t_cn);
    __syncthreads();
    print_vec(deleted_rows+offset_row[k], t_rn);
    __syncthreads();
    print_vec(results+offset_row[k], t_rn);
    __syncthreads();
    print_vec(row_group+offset_row[k], t_rn);
    __syncthreads();
    print_vec(col_group+offset_col[k], t_cn);
    __syncthreads();
    */

    for (search_depth = 1; search_depth <= vertex_num[k];) {
#ifndef BENCHMARK
      printf("search depth is %d\n", search_depth);
      // std::cout<<"deleted_cols "<<std::endl;
      // hipDeviceSynchronize();
      printf("deleted_cols\n");
      print_vec(t_deleted_cols, t_cn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("deleted_rows\n");
      print_vec(t_deleted_rows, t_rn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("results\n");
      print_vec(t_results, t_rn);
// hipDeviceSynchronize();
#endif

      t_existance_of_candidate_rows = 0;
      selected_row_id[k] = t_rn;
      t_conflict_node_id = 0;
      t_conflict_col_id = 0;
      t_conflict_edge[0] = 0;
      t_conflict_edge[1] = 0;
      // existance_of_candidate_rows=0;
      // selected_row_id=-1;
      check_existance_of_candidate_rows(
          t_deleted_rows, t_row_group, search_depth,
          &t_existance_of_candidate_rows, selected_row_id + k, t_rn);
      __syncthreads();
      // printf()
      // hipMemcpy(existance_of_candidate_rows,
      // existance_of_candidate_rows_gpu, sizeof(int), hipMemcpyDeviceToHost);
      // std::cout<<"check_existance_of_candidate_rows "<<std::endl;
      if (t_existance_of_candidate_rows == 1) { // check if there are candidate
                                                 // rows existing, if no, do
                                                 // backtrace
// select_row <<<block_count, thread_count >>> (deleted_rows, row_group,
// search_depth, total_dl_matrix_row_num, selected_row_id_gpu); //select
// row and add to results
// hipMemcpy(selected_row_id, selected_row_id_gpu, sizeof(int),
// hipMemcpyDeviceToHost);
#ifndef BENCHMARK
        printf("selected row id is %d \n", selected_row_id[k]);
#endif
        //__syncthreads();
        // hipMemset(&results[*selected_row_id],search_depth,sizeof(int));
        t_results[selected_row_id[k]] = search_depth;
        // set_vector_value<<<1,1>>>(results, *selected_row_id, search_depth);
        delete_rows_and_columns(t_dl_matrix, t_next_row, t_next_col,
                                t_deleted_rows, t_deleted_cols, search_depth,
                                selected_row_id[k], t_rn,
                                t_cn); // delete covered rows and columns
        __syncthreads();
        // deleted_rows[*selected_row_id] = -search_depth;
        t_deleted_rows[selected_row_id[k]] = -search_depth;
        // set_vector_value<<<1,1>>>(deleted_rows, *selected_row_id,
        // -search_depth);

        search_depth++; // next step
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      } else { // do backtrace
        search_depth--;
        if (search_depth > 0) {

          get_conflict_node_id(t_deleted_rows, t_row_group, search_depth,
                               &t_conflict_node_id, t_rn);
          if (t_conflict_node_id > 0) {
            __syncthreads();
            get_conflict_edge(t_dl_matrix, t_deleted_rows, t_row_group,
                              t_conflict_node_id, search_depth,
                              t_conflict_edge, vertex_num[k], t_rn, t_cn);
            __syncthreads();
            get_conflict_col_id(t_dl_matrix, t_deleted_cols,
                                &t_conflict_col_id, t_conflict_edge, t_cn,
                                vertex_num[k]);
            __syncthreads();

            // conflict_count[*conflict_col_id]++;
            // update conflict edge count
            t_conflict_count[t_conflict_col_id]++;
            // add_gpu<<<1,1>>>(&deleted_rows[*selected_row_id],1);
            recover_deleted_rows(t_deleted_rows, search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(t_deleted_cols, search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(t_results, search_depth,
                            t_rn); // recover results
            //__syncthreads();
            // hipMemcpy(&current_conflict_count,
            // &conflict_count[*conflict_col_id], sizeof(int),
            // hipMemcpyDeviceToHost);
            if (t_conflict_count[t_conflict_col_id] >
                hard_conflict_threshold) {
              search_depth = 1;
              init_vectors(t_conflict_count, t_cn);
              init_vectors_reserved(t_deleted_cols, t_cn);
              init_vectors(t_deleted_rows, t_rn);
              init_vectors(t_results, t_rn);
              __syncthreads();
              remove_cols(t_deleted_cols, t_col_group, t_conflict_col_id,
                          t_cn);
              __syncthreads();
              t_deleted_cols[t_conflict_col_id] = -1;

              // /hipMemset(&deleted_cols[*conflict_col_id],-1,sizeof(int));
            }
          } else {
            recover_deleted_rows(t_deleted_rows, search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(t_deleted_cols, search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(t_results, search_depth,
                            t_rn); // recover results
          }
        } else { // if all vertices are gone through, directly remove the edge
                 // with largest conflict count.
          search_depth = 1;
          t_max=0;
          get_largest_value(t_conflict_count, t_cn, &t_max);

          // hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
          // hipMemcpyDeviceToHost);
          __syncthreads();
          find_index(t_conflict_count, t_cn, &t_max, &t_conflict_col_id);
          init_vectors(t_conflict_count, t_cn);
          init_vectors_reserved(t_deleted_cols, t_cn);
          init_vectors(t_deleted_rows, t_rn);
          init_vectors(t_results, t_rn);
          __syncthreads();
          remove_cols(t_deleted_cols, t_col_group, t_conflict_col_id, t_cn);
        }
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      }
    }
    __syncthreads();
    for(int i=threadIdx.x; i < t_rn; i+=blockDim.x){
      t_final_results[i] = t_results[i];
    }
  }
}

} // namespace gpu_mg

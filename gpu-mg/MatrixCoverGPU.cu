#include "hip/hip_runtime.h"
#include "MatrixCoverGPU.cuh"

namespace gpu_mg {

constexpr int size_bit = 1 << 31;

__device__ void delete_rows_and_columns(
    const bool *dl_matrix, const int *next_row, const int *next_col,
    int *deleted_rows, int *deleted_cols, const int search_depth,
    const int selected_row_id, const int total_dl_matrix_row_num,
    const int total_dl_matrix_col_num) {
  int selected_row_idx = selected_row_id * total_dl_matrix_col_num;

  for (int i = threadIdx.x; i < total_dl_matrix_col_num;
       // // The below line will have negative effect of the col number is small
       //  i += (next_col[selected_row_idx + i] + blockDim.x - 1) / blockDim.x
       i += blockDim.x) {
    if (deleted_cols[i] == 0 && dl_matrix[selected_row_idx + i]) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num;
           j += next_row[i * total_dl_matrix_row_num + j]) {
        if (deleted_rows[j] == 0 &&
            dl_matrix[j * total_dl_matrix_col_num + i]) {
          deleted_rows[j] = search_depth;
        }
      }
    }
  }
  __syncthreads();
}

__device__ void init_vectors(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/

__device__ void get_largest_value(int *vec, const int vec_length, int *max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {

    atomicMax(max, vec[i]);
  }
}

__device__ void find_index(int *vec, const int vec_length, int *value,
                           int *index) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == *value) {
      atomicMax(index, i);
    }
  }
}

__device__ void init_vectors_reserved(int *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    // if (vec[i] != -1) {
    vec[i] &= size_bit;
    // }
  }
}

__device__ void check_existance_of_candidate_rows(
    int *deleted_rows, int *row_group, const int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i<total_dl_matrix_row_num && * selected_row_id> i;
       i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      // atomicExch(token, 1);
      *token = 1;
      atomicMin(selected_row_id, i);
      // If find a number can break;
      break;
    }
  }
  __syncthreads();
}

__device__ void get_vertex_row_group(int *row_group, const bool *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] +=
          static_cast<int>(dl_matrix[i * total_dl_matrix_col_num + j]) *
          (j + 1);
    }
  }
}

/*
__device__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__device__ void recover_deleted_rows(int *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__device__ void recover_deleted_cols(int *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__device__ void recover_results(int *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__device__ void get_conflict_node_id(int *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] < search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_edge(int *deleted_rows, int *row_group,
                                  const int conflict_node_id,
                                  const int search_depth, int *conflict_edge,
                                  const int vertex_num,
                                  const int total_dl_matrix_row_num,
                                  const int total_dl_matrix_col_num) {
  //*conflict_col_id = 0;
  // int idxa = 0;
  // int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // find the conflict edge that connects current node and the most closest
    // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    }
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge + 1, i);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_col_id(const bool *dl_matrix, int *deleted_cols,
                                    int *conflict_col_id, int *conflict_edge,
                                    int total_dl_matrix_col_num,
                                    int vertex_num) {
  // if(threadIdx.x==0){
  //  printf("conflict edge a %d edge b
  //  %d\n",conflict_edge[0],conflict_edge[1]);
  // }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num; j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] ==
            dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
        deleted_cols[j] > 0 &&
        dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j]) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__device__ void remove_cols(int *deleted_cols, int *col_group,
                            const int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = size_bit;
    }
  }
}

__device__ void print_vec(int *vec, int vec_length) {
  for (int i = 0; i < vec_length; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
}

/*
__global__ inline void print_vec_g(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}

*/
__device__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__device__ void add_gpu(int *device_var, int val) {
  atomicAdd(device_var, val);
}

__device__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

__global__ void
mc_solver(int *dl_matrix, int *next_col, int *next_row, int *results,
          int *deleted_cols, int *deleted_rows, int *col_group, int *row_group,
          int *conflict_count, int *vertex_num, int *total_dl_matrix_row_num,
          int *total_dl_matrix_col_num, int *offset_col, int *offset_row,
          int *offset_matrix, int *search_depth, int *selected_row_id,
          int *current_conflict_count, int *conflict_node_id,
          int *conflict_col_id, int *existance_of_candidate_rows,
          int *conflict_edge, int *max, const int graph_count,
          const int hard_conflict_threshold) {
  // // The Max Col Cnt is 131
  // // The Max Row Cnt is 339
  __shared__ bool shared_dl_matrix[140 * 340];

  // int k = blockDim.x;
  for (int k = blockIdx.x; k < graph_count; k += gridDim.x) {
    int t_cn = total_dl_matrix_col_num[k];
    int t_rn = total_dl_matrix_row_num[k];
    int *t_conflict_count = conflict_count + offset_col[k];
    int *t_deleted_cols = deleted_cols + offset_col[k];
    int *t_deleted_rows = deleted_rows + offset_row[k];
    int *t_results = results + offset_row[k];
    int *t_row_group = row_group + offset_row[k];
    int *t_col_group = col_group + offset_col[k];
    int *t_dl_matrix = dl_matrix + offset_matrix[k];
    int *t_next_col = next_col + offset_matrix[k];
    int *t_next_row = next_row + offset_matrix[k];
    int *t_conflict_edge = conflict_edge + 2 * k;

    int cxr = t_cn * t_rn;
    for (int i = threadIdx.x; i < cxr; i += blockDim.x) {
      shared_dl_matrix[i] = t_dl_matrix[i] == 1;
    }
    __syncthreads();

#ifndef BENCHMARK
    printf("blockID is %d\n", k);
    printf("vertexnum is %d\n", vertex_num[k]);
    printf("init conflict count \n");
#endif
    init_vectors(t_conflict_count, t_cn);
#ifndef BENCHMARK
    for (int i = 0; i < t_cn; i++) {
      printf("%d ", t_conflict_count[i]);
    }
    printf("\n");
#endif
    init_vectors(t_deleted_cols, t_cn);
    init_vectors(t_deleted_rows, t_rn);
    init_vectors(t_results, t_rn);
    __syncthreads();
    get_vertex_row_group(t_row_group, shared_dl_matrix, vertex_num[k], t_rn,
                         t_cn);
    __syncthreads();
    /*
    print_vec(deleted_cols+offset_col[k], t_cn);
    __syncthreads();
    print_vec(deleted_rows+offset_row[k], t_rn);
    __syncthreads();
    print_vec(results+offset_row[k], t_rn);
    __syncthreads();
    print_vec(row_group+offset_row[k], t_rn);
    __syncthreads();
    print_vec(col_group+offset_col[k], t_cn);
    __syncthreads();
    */

    for (search_depth[k] = 1; search_depth[k] <= vertex_num[k];) {
#ifndef BENCHMARK
      printf("search depth is %d\n", search_depth[k]);
      printf("deleted_cols\n");
      print_vec(t_deleted_cols, t_cn);
      printf("deleted_rows\n");
      print_vec(t_deleted_rows, t_rn);
      printf("results\n");
      print_vec(t_results, t_rn);
#endif

      existance_of_candidate_rows[k] = 0;
      selected_row_id[k] = t_rn;
      conflict_node_id[k] = 0;
      conflict_col_id[k] = 0;
      t_conflict_edge[0] = 0;
      t_conflict_edge[1] = 0;
      // existance_of_candidate_rows=0;
      // selected_row_id=-1;
      check_existance_of_candidate_rows(
          t_deleted_rows, t_row_group, search_depth[k],
          existance_of_candidate_rows + k, selected_row_id + k, t_rn);
      __syncthreads();
      if (existance_of_candidate_rows[k] == 1) { // check if there are candidate
                                                 // rows existing, if no, do
                                                 // backtrace
#ifndef BENCHMARK
        printf("selected row id is %d \n", selected_row_id[k]);
#endif
        //__syncthreads();
        t_results[selected_row_id[k]] = search_depth[k];
        // set_vector_value<<<1,1>>>(results, *selected_row_id, search_depth);
        delete_rows_and_columns(shared_dl_matrix, t_next_row, t_next_col,
                                t_deleted_rows, t_deleted_cols, search_depth[k],
                                selected_row_id[k], t_rn,
                                t_cn); // delete covered rows and columns
        __syncthreads();
        t_deleted_rows[selected_row_id[k]] = -search_depth[k];

        search_depth[k]++; // next step
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      } else { // do backtrace
        search_depth[k]--;
        if (search_depth[k] > 0) {

          get_conflict_node_id(t_deleted_rows, t_row_group, search_depth[k],
                               conflict_node_id + k, t_rn);
          if (conflict_node_id[k] > 0) {
            __syncthreads();
            get_conflict_edge(t_deleted_rows, t_row_group, conflict_node_id[k],
                              search_depth[k], t_conflict_edge, vertex_num[k],
                              t_rn, t_cn);
            __syncthreads();
            get_conflict_col_id(shared_dl_matrix, t_deleted_cols,
                                conflict_col_id + k, t_conflict_edge, t_cn,
                                vertex_num[k]);
            __syncthreads();
            // update conflict edge count
            t_conflict_count[conflict_col_id[k]]++;
            recover_deleted_rows(t_deleted_rows, search_depth[k],
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(t_deleted_cols, search_depth[k],
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(t_results, search_depth[k],
                            t_rn); // recover results
            //__syncthreads();
            if (t_conflict_count[conflict_col_id[k]] >
                hard_conflict_threshold) {
              search_depth[k] = 1;
              init_vectors(t_conflict_count, t_cn);
              init_vectors_reserved(t_deleted_cols, t_cn);
              init_vectors(t_deleted_rows, t_rn);
              init_vectors(t_results, t_rn);
              __syncthreads();
              remove_cols(t_deleted_cols, t_col_group, conflict_col_id[k],
                          t_cn);
              __syncthreads();
              t_deleted_cols[conflict_col_id[k]] = size_bit;
            }
          } else {
            recover_deleted_rows(t_deleted_rows, search_depth[k],
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(t_deleted_cols, search_depth[k],
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(t_results, search_depth[k],
                            t_rn); // recover results
          }
        } else { // if all vertices are gone through, directly remove the edge
                 // with largest conflict count.
          search_depth[k] = 1;
          get_largest_value(t_conflict_count, t_cn, max + k);

          // hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
          // hipMemcpyDeviceToHost);
          __syncthreads();
          find_index(t_conflict_count, t_cn, max + k, conflict_col_id + k);
          init_vectors(t_conflict_count, t_cn);
          init_vectors_reserved(t_deleted_cols, t_cn);
          init_vectors(t_deleted_rows, t_rn);
          init_vectors(t_results, t_rn);
          __syncthreads();
          remove_cols(t_deleted_cols, t_col_group, conflict_col_id[k], t_cn);
        }
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      }
    }
  }
}

} // namespace gpu_mg

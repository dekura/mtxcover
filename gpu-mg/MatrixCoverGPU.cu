#include "hip/hip_runtime.h"
#include "MatrixCoverGPU.cuh"

namespace gpu_mg {

constexpr int size_bit = 1 << 31;

__device__ void delete_rows_and_columns(
    const int *dl_matrix, const int *next_row, const int *next_col,
    short *deleted_rows, short *deleted_cols, const int search_depth,
    const int selected_row_id, const int total_dl_matrix_row_num,
    const int total_dl_matrix_col_num) {
  int selected_row_idx = selected_row_id * total_dl_matrix_col_num;

  for (int i = threadIdx.x; i < total_dl_matrix_col_num;
       // // The below line will have negative effect of the col number is small
       //  i += (next_col[selected_row_idx + i] + blockDim.x - 1) / blockDim.x
       i += blockDim.x) {
    if (deleted_cols[i] == 0 && dl_matrix[selected_row_idx + i] == 1) {
      deleted_cols[i] = search_depth;
      for (int j = 0; j < total_dl_matrix_row_num;
           j += next_row[i * total_dl_matrix_row_num + j]) {
        if (deleted_rows[j] == 0 &&
            dl_matrix[j * total_dl_matrix_col_num + i] == 1) {
          deleted_rows[j] = search_depth;
        }
      }
    }
  }
  __syncthreads();
}

template <typename T> 
__device__ void init_vectors(T *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    vec[i] = 0;
  }
}

/*
void get_largest_value_launcher(int* vec, hipcub::KeyValuePair<int, int> *argmax,
int vec_length)
{
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, vec,
argmax, vec_length);
        hipFree(d_temp_storage);
}
*/



template <typename T>
__device__ void get_largest_value(T *vec, const int vec_length, int *max) {

  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {

    atomicMax(max, vec[i]);
  }
}


template <typename T>
__device__ void find_index(T *vec, const int vec_length, int *value,
                           int *index) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    if (vec[i] == *value) {
      atomicMax(index, i);
    }
  }
}


template <typename T>
__device__ void init_vectors_reserved(T *vec, const int vec_length) {
  for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
    // if (vec[i] != -1) {
    vec[i] &= size_bit;
    // }
  }
}

__device__ void check_existance_of_candidate_rows(
    short *deleted_rows, const int *row_group, int search_depth, int *token,
    int *selected_row_id, const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i<total_dl_matrix_row_num && * selected_row_id> i;
       i = i + blockDim.x) {
    // std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
    if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
      // std::cout<<"Candidate Row Found...."<<std::endl;
      // atomicExch(token, 1);
      *token = 1;
      atomicMin(selected_row_id, i);
      // If find a number can break;
      break;
    }
  }
  __syncthreads();
}

__device__ void get_vertex_row_group(int *row_group, int *dl_matrix,
                                     const int vertex_num,
                                     const int total_dl_matrix_row_num,
                                     const int total_dl_matrix_col_num) {
  // printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    for (int j = 0; j < vertex_num; j++) {
      row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
    }
  }
}

/*
__device__ void select_row(int* deleted_rows, int* row_group, const int
search_depth, const int total_dl_matrix_row_num, int* selected_row_id)
{
        for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i +
blockDim.x)
        {
                if (deleted_rows[i] == 0 && row_group[i] == search_depth)
                {
                        atomicExch(selected_row_id, i);
                        atomicMin(selected_row_id, i);
                }
        }
        __syncthreads();
}
*/

__device__ void recover_deleted_rows(short *deleted_rows, const int search_depth,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (abs(deleted_rows[i]) > search_depth ||
        deleted_rows[i] == search_depth) {
      deleted_rows[i] = 0;
    }
  }
}

__device__ void recover_deleted_cols(short *deleted_cols, const int search_depth,
                                     const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (deleted_cols[i] >= search_depth) {
      deleted_cols[i] = 0;
    }
  }
}

__device__ void recover_results(short *results, const int search_depth,
                                const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (results[i] == search_depth) {
      results[i] = 0;
    }
  }
}

// problem: need to optimized to map on GPU array
__device__ void get_conflict_node_id(short *deleted_rows, int *row_group,
                                     const int search_depth,
                                     int *conflict_node_id,
                                     const int total_dl_matrix_row_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] < search_depth + 1) {
      atomicMax(conflict_node_id, deleted_rows[i]);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_edge(const int *dl_matrix, short *deleted_rows,
                                  const int *row_group, int conflict_node_id,
                                  int search_depth, int *conflict_edge,
                                  const int vertex_num,
                                  const int total_dl_matrix_row_num,
                                  const int total_dl_matrix_col_num) {
  //*conflict_col_id = 0;
  // int idxa = 0;
  // int idxb = 0;

  for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
    // find the conflict edge that connects current node and the most closest
    // node.
    if (deleted_rows[i] == -conflict_node_id) {
      atomicMax(conflict_edge, i);
    }
    if (row_group[i] == search_depth + 1 &&
        deleted_rows[i] == conflict_node_id) {
      atomicMax(conflict_edge + 1, i);
    }
  }
  __syncthreads();
}

__device__ void get_conflict_col_id(const int *dl_matrix, short *deleted_cols,
                                    int *conflict_col_id, int *conflict_edge,
                                    int total_dl_matrix_col_num,
                                    int vertex_num) {
  // if(threadIdx.x==0){
  //  printf("conflict edge a %d edge b
  //  %d\n",conflict_edge[0],conflict_edge[1]);
  // }
  for (int j = threadIdx.x; j < total_dl_matrix_col_num; j = j + blockDim.x) {
    if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j] ==
            dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
        deleted_cols[j] > 0 &&
        dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] == 1) {
      atomicMax(conflict_col_id, j);
    }
  }
  __syncthreads();
}

__device__ void remove_cols(short *deleted_cols, const int *col_group,
                            int conflict_col_id,
                            const int total_dl_matrix_col_num) {
  for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
    if (col_group[i] == col_group[conflict_col_id]) {
      deleted_cols[i] = size_bit;
    }
  }
}

__device__ void print_vec(int *vec, int vec_length) {
  for (int i = 0; i < vec_length; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
}

/*
__global__ inline void print_vec_g(int *vec, int vec_length)
{
        for(int i=0; i<vec_length; i++)
        {
                printf("%d ", vec[i]);
        }
        printf("\n");
}

*/
__device__ void print_mat(int *mat[], int total_dl_matrix_row_num,
                          int total_dl_matrix_col_num) {
  for (int i = 0; i < total_dl_matrix_row_num; i++) {
    for (int j = 0; j < total_dl_matrix_col_num; j++) {
      printf("%d ", mat[i][j]);
    }
    printf("\n");
  }
}

__device__ void add_gpu(int *device_var, int val) {
  atomicAdd(device_var, val);
}

__device__ void set_vector_value(int *device_var, int idx, int val) {
  device_var[idx] = val;
}

__global__ void 
init_vertex_group(int *row_group, int *dl_matrix, int* vertex_num, int* t_cn, int* t_rn, int *offset_row, int *offset_matrix, int graph_count) {
    int k=blockIdx.x;
    if(k<graph_count){
        get_vertex_row_group(row_group+offset_row[k], dl_matrix+offset_matrix[k], vertex_num[k], t_rn[k], t_cn[k]);
    }

}

__global__ void
mc_solver(int *dl_matrix, int *next_col, int *next_row, int *all_results,
          int *all_deleted_cols, int *all_deleted_rows, 
          int *col_group, int *row_group,
          int *all_conflict_count, int *vertex_num, int *total_dl_matrix_row_num,
          int *total_dl_matrix_col_num, int *offset_col, int *offset_row,
          int *offset_matrix, int *all_search_depth, int *all_selected_row_id,
          int *current_conflict_count, int *all_conflict_node_id,
          int *all_conflict_col_id, int *all_existance_of_candidate_rows,
          int *all_conflict_edge, int *all_max,const int graph_count,
          const int hard_conflict_threshold) {

  // to be refreshed if one conflict reaches many counts
  /*
  int search_depth = 0;
  int *selected_row_id_gpu;
  int vertex_num = vertex_num_gpu;
  int total_dl_matrix_col_num=total_dl_matrix_col_num_gpu;
  int total_dl_matrix_row_num=total_dl_matrix_row_num_gpu;
  int current_conflict_count;
  int *conflict_node_id_gpu;
  int *conflict_col_id_gpu;
  const int hard_conflict_threshold = 500;
  int *existance_of_candidate_rows_gpu;
  int *existance_of_candidate_rows=new int(0);
  int *conflict_col_id=new int(0);
  int *selected_row_id=new int(0);
  int *conflict_node_id=new int(0);
  hipMalloc(&existance_of_candidate_rows_gpu, sizeof(int));
  hipMalloc(&selected_row_id_gpu, sizeof(int));
  hipMalloc(&conflict_node_id_gpu, sizeof(int));
  hipMalloc(&conflict_col_id_gpu, sizeof(int));

  char brk;
  */
  // int k = blockDim.x;
  __shared__ short deleted_cols[128];
  __shared__ short deleted_rows[256];
  __shared__ short results[256];
  __shared__ unsigned short conflict_count[128];
  __shared__ int conflict_edge[2];
  __shared__ int existance_of_candidate_rows;
  __shared__ int selected_row_id;
  __shared__ int conflict_node_id;
  __shared__ int conflict_col_id;
  __shared__ int search_depth;
  int max = 0;
  int k = blockIdx.x; 
  if (k < graph_count) {
    const int t_cn = total_dl_matrix_col_num[k];
    const int t_rn = total_dl_matrix_row_num[k];
    //int *t_conflict_count = conflict_count + offset_col[k];
    //int *t_deleted_cols = deleted_cols + offset_col[k];
    //int *t_deleted_rows = deleted_rows + offset_row[k];
    int *t_results = all_results + offset_row[k];
    const int *t_row_group = row_group + offset_row[k];
    const int *t_col_group = col_group + offset_col[k];
    const int *t_dl_matrix = dl_matrix + offset_matrix[k];
    const int *t_next_col = next_col + offset_matrix[k];
    const int *t_next_row = next_row + offset_matrix[k];
    //int *t_conflict_edge = conflict_edge + 2 * k;

#ifndef BENCHMARK
    printf("blockID is %d\n", k);
    printf("vertexnum is %d\n", vertex_num[k]);
    printf("init conflict count \n");
#endif
    init_vectors(conflict_count, t_cn);
#ifndef BENCHMARK
    for (int i = 0; i < t_cn; i++) {
      printf("%d ", t_conflict_count[i]);
    }
    printf("\n");
#endif
    init_vectors(deleted_cols, t_cn);
    init_vectors(deleted_rows, t_rn);
    init_vectors(results, t_rn);
    //__syncthreads();
    //get_vertex_row_group(t_row_group, t_dl_matrix, vertex_num[k], t_rn, t_cn);
    __syncthreads();
    /*
    print_vec(deleted_cols+offset_col[k], t_cn);
    __syncthreads();
    print_vec(deleted_rows+offset_row[k], t_rn);
    __syncthreads();
    print_vec(results+offset_row[k], t_rn);
    __syncthreads();
    print_vec(row_group+offset_row[k], t_rn);
    __syncthreads();
    print_vec(col_group+offset_col[k], t_cn);
    __syncthreads();
    */

    for (search_depth = 1; search_depth <= vertex_num[k];) {
#ifndef BENCHMARK
      printf("search depth is %d\n", search_depth[k]);
      // std::cout<<"deleted_cols "<<std::endl;
      // hipDeviceSynchronize();
      printf("deleted_cols\n");
      print_vec(deleted_cols, t_cn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("deleted_rows\n");
      print_vec(deleted_rows, t_rn);
      // hipDeviceSynchronize();
      // hipDeviceSynchronize();
      printf("results\n");
      print_vec(results, t_rn);
// hipDeviceSynchronize();
#endif

      existance_of_candidate_rows = 0;
      selected_row_id = t_rn;
      conflict_node_id = 0;
      conflict_col_id = 0;
      conflict_edge[0] = 0;
      conflict_edge[1] = 0;
      // existance_of_candidate_rows=0;
      // selected_row_id=-1;
      check_existance_of_candidate_rows(
          deleted_rows, t_row_group, search_depth,
          &existance_of_candidate_rows, &selected_row_id, t_rn);
      __syncthreads();
      // printf()
      // hipMemcpy(existance_of_candidate_rows,
      // existance_of_candidate_rows_gpu, sizeof(int), hipMemcpyDeviceToHost);
      // std::cout<<"check_existance_of_candidate_rows "<<std::endl;
      if (existance_of_candidate_rows == 1) { // check if there are candidate
                                                 // rows existing, if no, do
                                                 // backtrace
// select_row <<<block_count, thread_count >>> (deleted_rows, row_group,
// search_depth, total_dl_matrix_row_num, selected_row_id_gpu); //select
// row and add to results
// hipMemcpy(selected_row_id, selected_row_id_gpu, sizeof(int),
// hipMemcpyDeviceToHost);
#ifndef BENCHMARK
        printf("selected row id is %d \n", selected_row_id);
#endif
        //__syncthreads();
        // hipMemset(&results[*selected_row_id],search_depth,sizeof(int));
        results[selected_row_id] = search_depth;
        // set_vector_value<<<1,1>>>(results, *selected_row_id, search_depth);
        delete_rows_and_columns(t_dl_matrix, t_next_row, t_next_col,
                                deleted_rows, deleted_cols, search_depth,
                                selected_row_id, t_rn,
                                t_cn); // delete covered rows and columns
        __syncthreads();
        // deleted_rows[*selected_row_id] = -search_depth;
        deleted_rows[selected_row_id] = -search_depth;
        // set_vector_value<<<1,1>>>(deleted_rows, *selected_row_id,
        // -search_depth);

        search_depth++; // next step
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      } else { // do backtrace
        search_depth--;
        if (search_depth > 0) {

          get_conflict_node_id(deleted_rows, row_group, search_depth,
                               &conflict_node_id, t_rn);
          if (conflict_node_id > 0) {
            __syncthreads();
            get_conflict_edge(t_dl_matrix, deleted_rows, t_row_group,
                              conflict_node_id, search_depth,
                              conflict_edge, vertex_num[k], t_rn, t_cn);
            __syncthreads();
            get_conflict_col_id(t_dl_matrix, deleted_cols,
                                &conflict_col_id, conflict_edge, t_cn,
                                vertex_num[k]);
            __syncthreads();

            // conflict_count[*conflict_col_id]++;
            // update conflict edge count
            conflict_count[conflict_col_id]++;
            // add_gpu<<<1,1>>>(&deleted_rows[*selected_row_id],1);
            recover_deleted_rows(deleted_rows, search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(deleted_cols, search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(results, search_depth,
                            t_rn); // recover results
            //__syncthreads();
            // hipMemcpy(&current_conflict_count,
            // &conflict_count[*conflict_col_id], sizeof(int),
            // hipMemcpyDeviceToHost);
            if (conflict_count[conflict_col_id] >
                hard_conflict_threshold) {
              search_depth = 1;
              init_vectors(conflict_count, t_cn);
              init_vectors_reserved(deleted_cols, t_cn);
              init_vectors(deleted_rows, t_rn);
              init_vectors(results, t_rn);
              __syncthreads();
              remove_cols(deleted_cols, col_group, conflict_col_id,
                          t_cn);
              __syncthreads();
              deleted_cols[conflict_col_id] = size_bit;

              // /hipMemset(&deleted_cols[*conflict_col_id],-1,sizeof(int));
            }
          } else {
            recover_deleted_rows(deleted_rows, search_depth,
                                 t_rn); // recover deleted
                                        // rows  previously
                                        // selected rows
            __syncthreads();
            recover_deleted_cols(deleted_cols, search_depth,
                                 t_cn); // recover deleted
                                        // cols except
                                        // afftected by
                                        // previously
                                        // selected rows
            __syncthreads();
            recover_results(results, search_depth,
                            t_rn); // recover results
          }
        } else { // if all vertices are gone through, directly remove the edge
                 // with largest conflict count.
          search_depth = 1;
          max = 0;
          get_largest_value(conflict_count, t_cn, &max);

          // hipMemcpy(conflict_col_id, conflict_col_id_gpu, sizeof(int),
          // hipMemcpyDeviceToHost);
          __syncthreads();
          find_index(conflict_count, t_cn, &max, &conflict_col_id);
          init_vectors(conflict_count, t_cn);
          init_vectors_reserved(deleted_cols, t_cn);
          init_vectors(deleted_rows, t_rn);
          init_vectors(results, t_rn);
          __syncthreads();
          remove_cols(deleted_cols, t_col_group, conflict_col_id, t_cn);
        }
        // print_vec(deleted_cols, total_dl_matrix_col_num);
        // print_vec(deleted_rows, total_dl_matrix_row_num);
        // print_vec(conflict_count, total_dl_matrix_col_num);
        // print_vec(results, total_dl_matrix_row_num);
      }
    }
    __syncthreads();
    for(int i=threadIdx.x; i < t_rn; i+=blockDim.x){
        t_results[i]=results[i];
        printf("result is %d", results[i]);
    }
  }
}

} // namespace gpu_mg
